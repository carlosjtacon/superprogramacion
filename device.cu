#include "hip/hip_runtime.h"
/**
 * All code related to device, such as kernell and data hanling (copying, 
 * freeing) is placed here.
 */
#include "device.h"

/**
* kernel no optimizado
**/
__global__ 
void generate_gpu(int* _old, int* _new, int w, int h)
 {
 	int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    int pos = i * w + j;

    struct offset moves[8];
    moves[0].i = -1;    moves[0].j = -1;
    moves[1].i = -1;    moves[1].j =  0;
    moves[2].i = -1;    moves[2].j =  1;
    moves[3].i =  0;    moves[3].j = -1;
    moves[4].i =  0;    moves[4].j =  1;
    moves[5].i =  1;    moves[5].j = -1;
    moves[6].i =  1;    moves[6].j =  0;
    moves[7].i =  1;    moves[7].j =  1;

 	int count = 0;
 	for (int m = 0; m < 8; ++m)
 	{
		// index of _old vector neigbour:
 		int old_p = ((i+moves[m].i)%w)*w + (j+moves[m].j)%h;
 		if (_old[old_p]>0)
 			count++;
 	}
 	if (count > 3 || count < 2){
 		_new[pos]=0;
 	}
 	else if (count == 2){
 		_new[pos] = _old[pos];
 	}
	else{				//count == 3
		_new[pos]=1;
	} 
}

/**
* kernel optimizado
**/
__global__ 
void generate_gpu_optimized(int* _old, int* _new, int w, int h)
 {
 	//coordenadas del hilo actual en _old o _new
 	int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    //posición en el array lineal de la celda actual
    int pos = i * w + j;
    //submatriz a ser leída, en memoria compartida, incluye vecinos exteriores
    __shared__ int sub_world [TILE_W+2] [TILE_H+2];
    //coordenadas del hilo actual en matriz sub_world
    int si = threadIdx.y +1;
    int sj = threadIdx.x +1;
    //cada hilo carga su vecino superior izquierdo
    sub_world[si-1][sj-1] = _old[d_mod((i-1),h)*w+d_mod((j-1),w)];
    //cada hilo de la fila inferior carga el vecino izq y izquierdo inferior
    if (threadIdx.y == TILE_H-1)
    {
	    sub_world[si][sj-1] = _old[i*h+d_mod((j-1),w)];
	    sub_world[si+1][sj-1] = _old[d_mod((i+1),h)*w+d_mod((j-1),w)];
    }
    //cada hilo de la columna derecha se carga a si mismo y su vecino derecho
    if (threadIdx.x == TILE_W-1)
    {
    	sub_world[si][sj] = _old[pos];
    	sub_world[si][sj+1] = _old[i*w + d_mod((j+1),w)];
    }
    //el hilo de la esquina superior derecha carga su vecino superior y superior derecho
    if (threadIdx.y == 0 && threadIdx.x == TILE_W-1)
    {
    	sub_world[si-1][sj] = _old[d_mod((i-1),h)*w+j];
    	sub_world[si-1][sj+1] = _old[d_mod((i-1),h)*w+d_mod((j+1),w)];
    }
    //el hilo de la esquina inferior derecha carga su vecino inferior e inferior derecho
    if (threadIdx.y == TILE_H-1 && threadIdx.x == TILE_W-1)
    {
    	sub_world[si+1][sj] = _old[d_mod((i+1),h)*w + j];
    	sub_world[si+1][sj+1] = _old[d_mod((i+1),h)*w + d_mod((j+1),w)];
    }
    //esperar a que toda la submatriz esté cargada
	__syncthreads();
 	if (i >= w || j >= h)
	 	return;
	//sumar todos vecinos, así evitamos divergencia
 	int count = sub_world[si-1][sj-1]+sub_world[si-1][sj]+sub_world[si-1][sj+1]
 		+sub_world[si][sj-1]+sub_world[si][sj+1]
 		+sub_world[si+1][sj-1]+sub_world[si+1][sj]+sub_world[si+1][sj+1];
	//aplicar reglas
 	if (count > 3 || count < 2){
 		_new[pos]=0;
 	}
 	else if (count == 2){
 		_new[pos] = _old[pos];
 	}
	else{				//count == 3
		_new[pos]=1;
	}
	 _new[pos] = sub_world[si-1][sj-1];
	// _new[pos] = threadIdx.x;
}

/**
* wrapper para el kernel no optimizado
**/
void call_generate_gpu(int* _old, int* _new, int w, int h)
{
	size_t size = w*h*sizeof(int);
    int* d_old;
	hipMalloc((void **)&d_old,size);
	int* d_new;
	hipMalloc((void **)&d_new,size);
    hipMemcpy(d_old,_old,size,hipMemcpyHostToDevice);
    hipMemcpy(d_new,_new,size,hipMemcpyHostToDevice);
	dim3 gridSize(8,8);
	dim3 blockSize(8,8);
	generate_gpu <<<gridSize, blockSize>>> (d_old, d_new, w, h);
	hipMemcpy(_old, d_old, size, hipMemcpyDeviceToHost);
	hipMemcpy(_new, d_new, size, hipMemcpyDeviceToHost);
	hipFree((void **)&d_old);
	hipFree((void **)&d_new);
}

/**
* wrapper para el kernel optimizado
**/
void call_generate_gpu_optimized(int* _old, int* _new, int w, int h)
{
	size_t size = w*h*sizeof(int);
    int* d_old;
	hipMalloc((void **)&d_old,size);
 	cout << "llega" << endl;
	int* d_new;
	hipMalloc((void **)&d_new,size);
    hipMemcpy(d_old,_old,size,hipMemcpyHostToDevice);
    hipMemcpy(d_new,_new,size,hipMemcpyHostToDevice);
    int gridx = w/TILE_W;
    int gridy= h/TILE_H;
    if (w%TILE_W > 0)
    	gridx+=1;
    if (h%TILE_H > 0)
    	gridy+=1;
    cout << "gridx=" << gridx <<endl;
    cout << "gridy=" << gridy <<endl;
	dim3 gridSize(gridx, gridy);
	dim3 blockSize(TILE_W,TILE_H);
	generate_gpu_optimized <<<gridSize, blockSize>>> (d_old, d_new, w, h);
	hipMemcpy(_old, d_old, size, hipMemcpyDeviceToHost);
	hipMemcpy(_new, d_new, size, hipMemcpyDeviceToHost);
	hipFree(d_old);
	hipFree(d_new);
}

/**
* realiza la operación a mod b; A diferencia de '%' en C,
* esta función devuelve siempre el modulo positivo (-1 mod 5 = 4)
**/
__device__ 
int d_mod(int a, int b)
{
	if (a < 0)
		return b+a;
	else
		return a%b;
}