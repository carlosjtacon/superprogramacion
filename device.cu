#include "hip/hip_runtime.h"
/**
 * All code related to device, such as kernell and data hanling (copying, 
 * freeing) is placed here.
 */
#include "device.h"

/**
* kernel no optimizado
**/
__global__ 
void generate_gpu(int* _old, int* _new, int w, int h)
 {

 	int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    int pos = i * w + j;

    if (i >= h || j >= w) {
    	return;
    }

    struct offset moves[8];
    moves[0].i = -1;    moves[0].j = -1;
    moves[1].i = -1;    moves[1].j =  0;
    moves[2].i = -1;    moves[2].j =  1;
    moves[3].i =  0;    moves[3].j = -1;
    moves[4].i =  0;    moves[4].j =  1;
    moves[5].i =  1;    moves[5].j = -1;
    moves[6].i =  1;    moves[6].j =  0;
    moves[7].i =  1;    moves[7].j =  1;

 	int count = 0;
 	for (int m = 0; m < 8; ++m)
 	{
		// index of _old vector neigbour:
 		int old_p = d_mod((i+moves[m].i), h)*w + d_mod((j+moves[m].j), w);
 		if (_old[old_p]>0)
 			count++;
 	}
 	if (count > 3 || count < 2){
 		_new[pos]=0;
 	}
 	else if (count == 2){
 		_new[pos] = _old[pos];
 	}
	else{				//count == 3
		_new[pos]=1;
	} 
}

/**
* kernel optimizado
**/
__global__ 
void generate_gpu_optimized(int* _old, int* _new, int w, int h)
 {
 	//coordenadas del hilo actual en _old o _new
 	int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= w || j >= h)
	 	return;
    //posición en el array lineal de la celda actual
    int pos = i * w + j;
    //submatriz a ser leída, en memoria compartida, incluye vecinos exteriores
    __shared__ int sub_world [TILE_W+2] [TILE_H+2];
    //coordenadas del hilo actual en matriz sub_world
    int si = threadIdx.y +1;
    int sj = threadIdx.x +1;
    //cada hilo se carga a si mismo
    sub_world[si][sj] = _old[pos];
	//esquina superior izquierda
	if (threadIdx.y == 0 && threadIdx.x == 0)
		sub_world[si-1][sj-1] = _old[d_mod(i-1,h)*w + d_mod(j-1,w)];
	//esquina superior derecha
	if (threadIdx.y == 0 && (threadIdx.x == TILE_W-1 || threadIdx.x == w-1))
		sub_world[si-1][sj+1] = _old[d_mod(i-1,h)*w + d_mod(j+1,w)];
	//esquina inferior izquierda
	if ((threadIdx.y == TILE_H-1 || threadIdx.y == h-1) && threadIdx.x == 0)
		sub_world[si+1][sj-1] = _old[d_mod(i+1,h)*w + d_mod(j-1,w)];
	//esquina inferior derecha
	if ((threadIdx.y == TILE_H-1 || threadIdx.y == h-1) && (threadIdx.x == TILE_W-1 ||threadIdx.x == w-1))
		sub_world[si+1][sj+1] = _old[d_mod(i+1,h)*w + d_mod(j+1,w)];
	//fila superior
	if (threadIdx.y == 0)
		sub_world[si-1][sj] = _old[d_mod(i-1,h)*w +j];
	//fila inferior
	if (threadIdx.y == TILE_H-1 || threadIdx.y == h-1)
		sub_world[si+1][sj] = _old[d_mod(i+1,h)*w +j];
	//columna izquierda
	if (threadIdx.x == 0)
		sub_world[si][sj-1] = _old[i*w +d_mod(j-1,w)];
	//columna derecha
	if (threadIdx.x == TILE_W-1 || threadIdx.x == w-1)
		sub_world[si][sj+1] = _old[i*w +d_mod(j+1,w)];
    //esperar a que toda la submatriz esté cargada
	__syncthreads();
	//sumar todos vecinos, así evitamos divergencia
 	int count = sub_world[si-1][sj-1]+sub_world[si-1][sj]+sub_world[si-1][sj+1]
 		+sub_world[si][sj-1]+sub_world[si][sj+1]
 		+sub_world[si+1][sj-1]+sub_world[si+1][sj]+sub_world[si+1][sj+1];
	//aplicar reglas
 	if (count > 3 || count < 2){
 		_new[pos]=0;
 	}
 	else if (count == 2){
 		_new[pos] = _old[pos];
 	}
	else{				//count == 3
		_new[pos]=1;
	}
	 // _new[pos] = sub_world[si-1][sj-1];
	// _new[pos] = threadIdx.x;
}

/**
* wrapper para el kernel no optimizado
**/
void call_generate_gpu(int* _old, int* _new, int w, int h)
{
	size_t size = w*h*sizeof(int);
 	int* d_old;
	hipMalloc((void **)&d_old,size);
	int* d_new;
	hipMalloc((void **)&d_new,size);
 	hipMemcpy(d_old,_old,size,hipMemcpyHostToDevice);
 	hipMemcpy(d_new,_new,size,hipMemcpyHostToDevice);
	
 	int wsize = w/16;
 	int hsize = h/16;
 	if(w%16 != 0) wsize++;
 	if(h%16 != 0) hsize++;

	dim3 blockSize(16, 16);
	dim3 gridSize(wsize, hsize);
	
	generate_gpu <<<gridSize, blockSize>>> (d_old, d_new, w, h);
	hipMemcpy(_old, d_old, size, hipMemcpyDeviceToHost);
	hipMemcpy(_new, d_new, size, hipMemcpyDeviceToHost);
	hipFree((void **)&d_old);
	hipFree((void **)&d_new);
}

/**
* wrapper para el kernel optimizado
**/
void call_generate_gpu_optimized(int* _old, int* _new, int w, int h)
{
	size_t size = w*h*sizeof(int);
    int* d_old;
	hipMalloc((void **)&d_old,size);
	int* d_new;
	hipMalloc((void **)&d_new,size);
    hipMemcpy(d_old,_old,size,hipMemcpyHostToDevice);
    hipMemcpy(d_new,_new,size,hipMemcpyHostToDevice);
    int gridx = w/TILE_W;
    int gridy= h/TILE_H;
    if (w%TILE_W > 0)
    	gridx+=1;
    if (h%TILE_H > 0)
    	gridy+=1;
	dim3 gridSize(gridx, gridy);
	dim3 blockSize(TILE_W,TILE_H);
	generate_gpu_optimized <<<gridSize, blockSize>>> (d_old, d_new, w, h);
	hipMemcpy(_old, d_old, size, hipMemcpyDeviceToHost);
	hipMemcpy(_new, d_new, size, hipMemcpyDeviceToHost);
	hipFree(d_old);
	hipFree(d_new);
}

/**
* realiza la operación a mod b; A diferencia de '%' en C,
* esta función devuelve siempre el modulo positivo (-1 mod 5 = 4)
**/
__device__ 
int d_mod(int a, int b)
{
	if (a < 0)
		return b+a;
	else
		return a%b;
}


