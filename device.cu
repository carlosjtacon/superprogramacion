#include "hip/hip_runtime.h"
/**
 * All code related to device, such as kernell and data hanling (copying, 
 * freeing) is placed here.
 */
#include "device.h"

/**
* kernel no optimizado
**/
__global__ 
void generate_gpu(int* _old, int* _new, int w, int h, offset moves[])
 {
 	int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    int pos = i * w + j;

    struct offset moves2[8];
    moves2[0].i = -1;    moves2[0].j = -1;
    moves2[1].i = -1;    moves2[1].j =  0;
    moves2[2].i = -1;    moves2[2].j =  1;
    moves2[3].i =  0;    moves2[3].j = -1;
    moves2[4].i =  0;    moves2[4].j =  1;
    moves2[5].i =  1;    moves2[5].j = -1;
    moves2[6].i =  1;    moves2[6].j =  0;
    moves2[7].i =  1;    moves2[7].j =  1;

 	int count = 0;
 	for (int m = 0; m < 8; ++m)
 	{
		// index of _old vector neigbour:
 		int old_p = ((i+moves2[m].i)%w)*w + (j+moves2[m].j)%h;
 		if (_old[old_p]>0)
 			count++;
 	}
 	if (count > 3 || count < 2){
 		_new[pos]=0;
 	}
 	else if (count == 2){
 		_new[pos] = _old[pos];
 	}
	else{				//count == 3
		_new[pos]=1;
	} 
}
/**
* wrapper para el kernel no optimizado
**/
void call_generate_gpu(int* _old, int* _new, int w, int h, offset moves[])
{
	size_t size = w*h*sizeof(int);
    int* d_old;
	hipMalloc((void **)&d_old,size);
	int* d_new;
	hipMalloc((void **)&d_new,size);
    hipMemcpy(d_old,_old,size,hipMemcpyHostToDevice);
    hipMemcpy(d_new,_new,size,hipMemcpyHostToDevice);
	/*nota mental: puede que pete porque no estoy reservando memoria para moves
	pero al pasarse como valor, no se si hace falta*/
	dim3 gridSize(8,8);
	dim3 blockSize(8,8);
	generate_gpu <<<gridSize, blockSize>>> (d_old, d_new, w, h, moves);
	hipMemcpy(_old, d_old, size, hipMemcpyDeviceToHost);
	hipMemcpy(_new, d_new, size, hipMemcpyDeviceToHost);
	hipFree((void **)&d_old);
	hipFree((void **)&d_new);
}
/**
* wrapper para el kernel optimizado
**/
void call_generate_gpu_optimized(int* _old, int* _new, int w, int h, offset moves[])
{
	size_t size = w*h*sizeof(int);
    int* d_old;
	hipMalloc((void **)&d_old,size);
 	cout << "llega" << endl;
	int* d_new;
	hipMalloc((void **)&d_new,size);
    hipMemcpy(d_old,_old,size,hipMemcpyHostToDevice);
    hipMemcpy(d_new,_new,size,hipMemcpyHostToDevice);
    int gridx = w/TILE_W;
    int gridy= h/TILE_H;
    if (w%TILE_W > 0)
    	gridx+=1;
    if (h%TILE_H > 0)
    	gridy+=1;
    cout << "gridx=" << gridx <<endl;
    cout << "gridy=" << gridy <<endl;
	dim3 gridSize(gridx, gridy);
	dim3 blockSize(TILE_W,TILE_H);
	generate_gpu_optimized <<<gridSize, blockSize>>> (d_old, d_new, w, h, moves);
	hipMemcpy(_old, d_old, size, hipMemcpyDeviceToHost);
	hipMemcpy(_new, d_new, size, hipMemcpyDeviceToHost);
	hipFree(d_old);
	hipFree(d_new);
}
/**
* realiza la operación a mod b; A diferencia de '%' en C,
* esta función devuelve siempre el modulo positivo (-1 mod 5 = 4)
**/
__device__ int mod(int a, int b)
{
	if (a < 0)
		return b+a;
	else
		return a%b;
}