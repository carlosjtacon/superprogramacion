#include "hip/hip_runtime.h"
/**
 * All code related to device, such as kernell and data hanling (copying, 
 * freeing) is placed here.
 */
#include "device.h"

__global__ 
void generate_gpu(int* _old, int* _new, int w, int h, offset moves[])
 {
 	int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    int pos = i * w + j;

    struct offset moves2[8];
    moves2[0].i = -1;    moves2[0].j = -1;
    moves2[1].i = -1;    moves2[1].j =  0;
    moves2[2].i = -1;    moves2[2].j =  1;
    moves2[3].i =  0;    moves2[3].j = -1;
    moves2[4].i =  0;    moves2[4].j =  1;
    moves2[5].i =  1;    moves2[5].j = -1;
    moves2[6].i =  1;    moves2[6].j =  0;
    moves2[7].i =  1;    moves2[7].j =  1;

 	int count = 0;
 	for (int m = 0; m < 8; ++m)
 	{
		// index of _old vector neigbour:
 		int old_p = ((i+moves2[m].i)%w)*w + (j+moves2[m].j)%h;
 		if (_old[old_p]>0)
 			count++;
 	}
 	if (count > 3 || count < 2){
 		_new[pos]=0;
 	}
 	else if (count == 2){
 		_new[pos] = _old[pos];
 	}
	else{				//count == 3
		_new[pos]=1;
	} 
}

void call_generate_gpu(int* _old, int* _new, int w, int h, offset moves[])
{
	size_t size = w*h*sizeof(int);
    int* d_old;
	hipMalloc((void **)&d_old,size);
	int* d_new;
	hipMalloc((void **)&d_new,size);
    hipMemcpy(d_old,_old,size,hipMemcpyHostToDevice);
    hipMemcpy(d_new,_new,size,hipMemcpyHostToDevice);
	/*nota mental: puede que pete porque no estoy reservando memoria para moves
	pero al pasarse como valor, no se si hace falta*/
	dim3 gridSize(8,8);
	dim3 blockSize(8,8);
	generate_gpu <<<gridSize, blockSize>>> (d_old, d_new, w, h, moves);
	hipMemcpy(_old, d_old, size, hipMemcpyDeviceToHost);
	hipMemcpy(_new, d_new, size, hipMemcpyDeviceToHost);
	hipFree((void **)&d_old);
	hipFree((void **)&d_new);
}